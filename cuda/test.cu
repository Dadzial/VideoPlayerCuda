#include <hip/hip_runtime.h>

__global__ void addKernel(int *a, int *b, int *c) {
    *c = *a + *b;
}

extern "C" void addCuda(int a, int b, int *c) {
    int *d_a, *d_b, *d_c;
    int h_c;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    addKernel<<<1,1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    *c = h_c;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
