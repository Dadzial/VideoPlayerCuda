#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(int a, int b, int *c) {
    if(c != nullptr) {
        *c = a + b;
    }
}


extern "C" int addNumbers() {
    int a = 5;
    int b = 7;
    int c = 0;

    int *d_c;
    hipMalloc(&d_c, sizeof(int));


    addKernel<<<1, 1>>>(a, b, d_c);


    hipDeviceSynchronize();


    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_c);

    return c;
}

